#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>

/*
Compile w ./a.out 1000 1000 asdf

*/

// Struct to get time
double gettime(void)
{
    struct timeval tval;
    gettimeofday(&tval, NULL);
    return ((double)tval.tv_sec + (double)tval.tv_usec / 10000000.0);
}

// Function for performing warp shuffle
__device__ double warp_shuffle(double var)
{
    // Gets every thread in warp
    unsigned mask = 0xffffffff;
    for (int diff = 32 / 2; diff > 0; diff = diff / 2)
    {
        var += __shfl_down_sync(mask, var, diff);
    }
    // Only lane 0 gets true value
    return var;
}

// Main conway function
__global__ void Conway(int *map1, int *map2, float *nChanges, int nRows, int thread_count, int sm_count, int remainder_for_last_block)
{

    // Value for just this row's changes, will be warp shuffled together later
    int local_changes = 0;

    // Get identification
    int blockNum = blockIdx.x;
    int threadNum = threadIdx.x;
    // int threadNumGlobal = blockIdx.x * blockDim.x + threadIdx.x;
    // printf("Hello from thread %d in block %d!\n", threadNum, blockNum);

    // Analogous to global thread id
    int my_row = blockNum * thread_count + threadNum;
    // printf("my row: %d\n", my_row);

    // This makes sure not to get the ghost row at the top and bottom
    if (my_row < nRows - 1 && my_row > 0)
    {

        // Each thread iterates over the duration of a row
        for (int j = 1; j < nRows - 1; j++)
        {
            // center = map1[my_row * nRows + j];
            // All neighbors
            int neighborCount = map1[my_row * nRows + j - 1] +
                                map1[my_row * nRows + j + 1] +
                                map1[(my_row - 1) * nRows + j - 1] +
                                map1[(my_row - 1) * nRows + j] +
                                map1[(my_row - 1) * nRows + j + 1] +
                                map1[(my_row + 1) * nRows + j - 1] +
                                map1[(my_row + 1) * nRows + j] +
                                map1[(my_row + 1) * nRows + j + 1];

            // GOL rules
            if (neighborCount == 3)
            {
                map2[my_row * nRows + j] = 1;
            }
            else if (neighborCount == 2)
            {
                map2[my_row * nRows + j] = map1[my_row * nRows + j];
            }
            else
            {
                map2[my_row * nRows + j] = 0;
            }


            // Keeps track of local changes
            if (map2[my_row * nRows + j] != map1[my_row * nRows + j])
            {
                local_changes++;
            }
        }
    }

    // Warp shuffles the local changes down to lane 0 and atomic adds
    local_changes = warp_shuffle(local_changes);
    int my_lane = threadIdx.x % 32;
    if (my_lane == 0)
    {
        atomicAdd(nChanges, local_changes);
    }

} /*Runs conway*/

int main(int argc, char *argv[])
{

    // Checks cmd args
    if (argc != 4)
    {
        printf("Usage: ./<executable(conway)> <board-size> <num-generations> <output_directory>\n");
        return 0;
    }

    int nRows = atoi(argv[1]) + 2;
    int nCols = nRows;
    int numGenerations = atoi(argv[2]);
    char *output_directory = argv[3];

    // Declares arrays
    int *map1;
    int *map2;
    hipMallocManaged(&map1, nRows * nCols * sizeof(int));
    hipMallocManaged(&map2, nRows * nCols * sizeof(int));

    // Fills initial state
    srand48(12345);
    for (int i = 0; i < nRows; i++)
    {

        for (int j = 0; j < nCols; j++)
        {

            if (i == 0 || i == nCols - 1 || j == 0 || j == nCols - 1)
            {
                map1[i * nRows + j] = 0;
                map2[i * nRows + j] = 0;
            }
            else
            {
                map1[i * nRows + j] = drand48() > 0.5 ? 1 : 0;
            }
        }
    }

    float *nChanges;
    hipMallocManaged(&nChanges, sizeof(float));
    *nChanges = 0;

    // Assume each thread block has 32 threads
    // This allows us to use warp-wide operations
    // Its gonna make the last case weird but so be it
    int thread_count = 32;
    int sm_count = (nRows) / 32;
    int remainder_for_last_block = (nRows) % 32;
    if (remainder_for_last_block > 0)
    {
        sm_count++;
    }
    // printf("%d\n", sm_count);
    // printf("%d\n", remainder_for_last_block);

    int gen = 0;

    double start = gettime();
    for (gen = 0; gen < numGenerations; gen++)
    {
        if (gen % 2 == 0)
        {
            Conway<<<sm_count, thread_count>>>(map1, map2, nChanges, nRows, thread_count, sm_count, remainder_for_last_block);
        }
        else
        {
            Conway<<<sm_count, thread_count>>>(map2, map1, nChanges, nRows, thread_count, sm_count, remainder_for_last_block);
        }
        hipDeviceSynchronize();

        if (*nChanges == 0)
        {
            printf("Exiting early due to no changes at generation %d\n", gen);
            break;
        }
    }
    double finish = gettime();
    printf("Finished Conway\n");

    printf("Test details:\n %d by %d board\n", nRows, nCols);
    printf("%d generations\n", numGenerations);
    printf("Time taken = %lf seconds\n", (finish - start));
    printf("Terminated at generation %d\n", gen);

    // printf("%d\n", *nChanges);

    // Writes results to the file
    FILE *file = fopen(output_directory, "w");
    if (file == NULL)
    {
        perror("Error opening file\n");
        return 1;
    }
    for (int i = 1; i < nRows - 1; i++)
    {
        for (int j = 1; j < nCols - 1; j++)
        {
            // printf("%d\n", i);
            if (gen % 2 == 0)
            {
                fprintf(file, "%d ", map1[i * nRows + j]);
            }
            else
            {
                fprintf(file, "%d ", map2[i * nRows + j]);
            }
        }

        fprintf(file, "\n"); // New line after each row
    }
    printf("Done\n");
    fclose(file);

    hipFree(map1);
    hipFree(map2);

    printf("Finished\n");

    return 0;
}
